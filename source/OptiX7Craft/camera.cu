#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>
#include <optix_device.h>
#include "OptiX7Craft.h"
#include "random.h"
#include "helpers.h"
#include <cuda/helpers.h>

extern "C" {
__constant__ Params params;
}

extern "C" __global__ void __raygen__pinhole_camera()
{
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    const CameraData* camera = (CameraData*) optixGetSbtDataPointer();

    const unsigned int image_index = params.width * idx.y + idx.x;
    unsigned int       seed        = tea<16>( image_index, params.subframe_index );
    float3 result = make_float3(0);

    int i = params.samples_per_launch; //spp
    do {
        float2 subpixel_jitter =  make_float2(rnd(seed), rnd(seed));

        float2 d = ((make_float2(idx.x, idx.y) + subpixel_jitter) / make_float2(params.width, params.height)) * 2.f - 1.f;
        float3 ray_origin = camera->eye;
        float3 ray_direction = normalize(d.x*camera->U + d.y*camera->V + camera->W);

        
        SunPRD sun_prd;
        sun_prd.importance = 1.f;
        sun_prd.depth = 0;
        sun_prd.seed = seed;
        sun_prd.done = false;
        sun_prd.attenuation = make_float3(1.0f);
        sun_prd.emitted = make_float3(0.0f);
        sun_prd.countEmitted = true;
        sun_prd.radiance = make_float3(0.0f);


        unsigned int u0, u1;
        packPointer(&sun_prd, u0, u1);
        optixTrace(
            params.handle,
            ray_origin,
            ray_direction,
            params.scene_epsilon,
            1e16f,
            0.0f,
            OptixVisibilityMask(1),
            OPTIX_RAY_FLAG_NONE,
            RAY_TYPE_RADIANCE,
            RAY_TYPE_COUNT,
            RAY_TYPE_RADIANCE,
            u0,
            u1);
        result += sun_prd.emitted;
        result += sun_prd.radiance * sun_prd.attenuation;

    }
    while (--i);

    float3         accum_color = result / static_cast<float>(params.samples_per_launch);
    if( params.subframe_index > 0 )
    {
        const float                 a = 1.0f / static_cast<float>(params.subframe_index + 1);
        const float3 accum_color_prev = make_float3(params.accum_buffer[image_index]);
        accum_color = lerp(accum_color_prev, accum_color, a);
    }

    params.accum_buffer[image_index] = make_float4(accum_color, 1.0f);
    params.frame_buffer[image_index] = make_color(accum_color);

}



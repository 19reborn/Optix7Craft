#include "hip/hip_runtime.h"
//
// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <hip/hip_vector_types.h>
#include <optix_device.h>
#include "optixWhitted.h"
#include "random.h"
#include "helpers.h"
#include <cuda/helpers.h>

extern "C" {
__constant__ Params params;
}

__inline__ __device__ float3 tonemap(const float3 in)
{
    // hard coded exposure for sun/sky
    const float exposure = 1.0f / 30.0f;
    float3 x = exposure * in;

    // "filmic" map from a GDC talk by John Hable.  This includes 1/gamma.
    x = fmaxf(x - make_float3(0.004f), make_float3(0.0f));
    float3 ret = (x * (6.2f * x + make_float3(.5f))) / (x * (6.2f * x + make_float3(1.7f)) + make_float3(0.06f));

    return ret;
}

extern "C" __global__ void __raygen__pinhole_camera()
{
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    const CameraData* camera = (CameraData*) optixGetSbtDataPointer();

    const unsigned int image_index = params.width * idx.y + idx.x;
    unsigned int       seed        = tea<16>( image_index, params.subframe_index );
    float3 result = make_float3(0);
    // Subpixel jitter: send the ray through a different position inside the pixel each time,
    // to provide antialiasing. The center of each pixel is at fraction (0.5,0.5)
    float2 subpixel_jitter = params.subframe_index == 0 ?
        make_float2(0.5f, 0.5f) : make_float2(rnd( seed ), rnd( seed ));

    float2 d = ((make_float2(idx.x, idx.y) + subpixel_jitter) / make_float2(params.width, params.height)) * 2.f - 1.f;
    float3 ray_origin = camera->eye;
    float3 ray_direction = normalize(d.x*camera->U + d.y*camera->V + camera->W);
    /*
    RadiancePRD prd;
    prd.importance = 1.f;
    prd.depth = 0;

    optixTrace(
        params.handle,
        ray_origin,
        ray_direction,
        params.scene_epsilon,
        1e16f,
        0.0f,
        OptixVisibilityMask( 1 ),
        OPTIX_RAY_FLAG_NONE,
        RAY_TYPE_RADIANCE,
        RAY_TYPE_COUNT,
        RAY_TYPE_RADIANCE,
        float3_as_args(prd.result),
        reinterpret_cast<unsigned int&>(prd.importance),
        reinterpret_cast<unsigned int&>(prd.depth) );

    result += prd.result;
    */
   
    SunPRD sun_prd;
    sun_prd.importance = 1.f;
    sun_prd.depth = 0;
    sun_prd.seed = seed;
    sun_prd.done = false;
    sun_prd.attenuation = make_float3(1.0f);

    // light from a light source or miss program
    sun_prd.radiance = make_float3(0.0f);
    // next ray to be traced
    sun_prd.origin = make_float3(0.0f);
    sun_prd.direction = make_float3(0.0f);

    //texture
    sun_prd.origin_dx = make_float3(0.0f);
    sun_prd.origin_dy = make_float3(0.0f);
    const float Wlen = length(camera->W);
    // TODO: This is not 100% correct, since U and V are not perpendicular to the ray direction
    sun_prd.direction_dx = camera->U * (2.0f / (params.width * Wlen));
    sun_prd.direction_dy = camera->V * (2.0f / (params.height * Wlen));

    for (;;) {
        unsigned int u0, u1;
        packPointer(&sun_prd, u0, u1);
        //optixSetPayload_0(u0);
        //optixSetPayload_1(u1);
        optixTrace(
            params.handle,
            ray_origin,
            ray_direction,
            params.scene_epsilon,
            1e16f,
            0.0f,
            OptixVisibilityMask(1),
            OPTIX_RAY_FLAG_NONE,
            RAY_TYPE_RADIANCE,
            RAY_TYPE_COUNT,
            RAY_TYPE_RADIANCE,
            u0,
            u1);

        result += sun_prd.radiance * sun_prd.attenuation;

        if (sun_prd.done) {
            break;
        }
        else if (sun_prd.depth >= 2) {
            result += sun_prd.attenuation * make_float3(0.2f,0.2f,0.2f);
            break;
        }

        sun_prd.depth++;

        // Update ray data for the next path segment
        ray_origin = sun_prd.origin;
        ray_direction = sun_prd.direction;
    }
    
    float4 acc_val = params.accum_buffer[image_index];
    if( params.subframe_index > 0 )
    {
        acc_val = lerp( acc_val, make_float4( result, 0.f), 1.0f / static_cast<float>( params.subframe_index+1 ) );
    }
    else
    {
        acc_val = make_float4(result, 0.f);
    }
    params.frame_buffer[image_index] = make_color(tonemap(make_float3(acc_val)));
    params.accum_buffer[image_index] = acc_val;

}


/*
static __forceinline__ __device__ RadiancePRD* getPRD()
{
    const unsigned int u0 = optixGetPayload_0();
    const unsigned int u1 = optixGetPayload_1();
    return reinterpret_cast<RadiancePRD*>(unpackPointer(u0, u1));
}
*/


static __forceinline__ __device__ void setPayloadOcclusion(bool occluded)
{
    optixSetPayload_0(static_cast<unsigned int>(occluded));
}

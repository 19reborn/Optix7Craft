#include "hip/hip_runtime.h"
//
// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <hip/hip_vector_types.h>
#include <optix_device.h>
#include "OptiX7Craft.h"
#include "random.h"
#include "helpers.h"
#include <cuda/helpers.h>

extern "C" {
__constant__ Params params;
}

extern "C" __global__ void __raygen__pinhole_camera()
{
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    const CameraData* camera = (CameraData*) optixGetSbtDataPointer();

    const unsigned int image_index = params.width * idx.y + idx.x;
    unsigned int       seed        = tea<16>( image_index, params.subframe_index );
    float3 result = make_float3(0);

    int i = params.samples_per_launch; //spp
    do {
        float2 subpixel_jitter =  make_float2(rnd(seed), rnd(seed));

        float2 d = ((make_float2(idx.x, idx.y) + subpixel_jitter) / make_float2(params.width, params.height)) * 2.f - 1.f;
        float3 ray_origin = camera->eye;
        float3 ray_direction = normalize(d.x*camera->U + d.y*camera->V + camera->W);

        
        SunPRD sun_prd;
        sun_prd.importance = 1.f;
        sun_prd.depth = 0;
        sun_prd.seed = seed;
        sun_prd.done = false;
        sun_prd.attenuation = make_float3(1.0f);
        sun_prd.emitted = make_float3(0.0f);
        sun_prd.countEmitted = true;
        sun_prd.radiance = make_float3(0.0f);


        unsigned int u0, u1;
        packPointer(&sun_prd, u0, u1);
        optixTrace(
            params.handle,
            ray_origin,
            ray_direction,
            params.scene_epsilon,
            1e16f,
            0.0f,
            OptixVisibilityMask(1),
            OPTIX_RAY_FLAG_NONE,
            RAY_TYPE_RADIANCE,
            RAY_TYPE_COUNT,
            RAY_TYPE_RADIANCE,
            u0,
            u1);
        result += sun_prd.emitted;
        result += sun_prd.radiance * sun_prd.attenuation;

    }
    while (--i);

    float3         accum_color = result / static_cast<float>(params.samples_per_launch);
    if( params.subframe_index > 0 )
    {
        const float                 a = 1.0f / static_cast<float>(params.subframe_index + 1);
        const float3 accum_color_prev = make_float3(params.accum_buffer[image_index]);
        accum_color = lerp(accum_color_prev, accum_color, a);
    }

    params.accum_buffer[image_index] = make_float4(accum_color, 1.0f);
    params.frame_buffer[image_index] = make_color(accum_color);

}


